#include "hip/hip_runtime.h"
#define MAINFILE
#include <iostream>
#include <fstream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "hip/hip_runtime_api.h"

#include "constants.h"
#include "initial_conditions.h"
#include "evol.h"
#include "diagnostics.h"

using namespace std;

int main(int argc, char *argv[]){
	real *h_particles, *d_particles; // array of particles: 1d position, cosine of pitch angle, lorentz factor, exit time
	bool newflag = 1,newflag_trajectories = 1;
	float *dw;
	int threads_per_block = 32;
	int nblocks = nparticles/threads_per_block;  // make sure nparticles is a multiple of 32

	float milli;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	energy_kev = (real *)malloc(nparticles*sizeof(real));
	potential = (real *)malloc(nparticles*sizeof(real));
	h_particles = (real *)malloc(nparticles*nfields*sizeof(real));
	
	checkCudaErrors( hipMalloc((void **)&energy_kev, nparticles*sizeof(real)) );
	checkCudaErrors( hipMalloc((void **)&potential, nparticles*sizeof(real)) );
	checkCudaErrors( hipMalloc((void **)&d_particles, nparticles*nfields*sizeof(real)) );
	checkCudaErrors( hipMalloc((void **)&dw, nparticles*nt*sizeof(float)) );
	initialise(h_particles);

	checkCudaErrors( hipMemcpy(d_particles,h_particles,nparticles*nfields*sizeof(real),hipMemcpyHostToDevice) );

	// random number generation
	
	hipEventRecord(start);
	
	hiprandGenerator_t gen;
	checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
	checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
	checkCudaErrors( hiprandGenerateNormal(gen, dw, nparticles*nt, 0.0f, 1.0f) );
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);
	
	printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
	        milli, nparticles*nt/(0.001*milli));

	// run particles
	hipEventRecord(start);
	move_particles<<<nblocks,threads_per_block>>>(d_particles,dw,energy_kev,potential);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);
	
	printf("kernel  execution time (ms): %f\n", milli);


	//// main time loop
	//for (int j = 0; j < nt; j++){
	//	//if (j%1 == 0) write_particle(particles,newflag_trajectories);
	//	//newflag_trajectories = 0;


	//	//if (isnan((double) particles[0])){
	//	//	cout << "position is nan. stopping" << endl;
	//	//	return 0;
	//	//}
	//	//if (j % (nt/100) == 0) printf("timestep %d of %d\n",j,nt);
	//}

	checkCudaErrors( hipMemcpy(h_particles,d_particles,nparticles*nfields*sizeof(real),hipMemcpyDeviceToHost) );
	write_particles(h_particles,newflag,argv[1]);
	cout << "size of particles array " << nparticles << endl;
	//for (int j = 0; j < nparticles; j++) printf("particle %d final energy %f position %f\n",j,(particles[nfields*j+2]-1)*511.0,particles[nfields*j]*Lscl);
	
	free(h_particles);
	checkCudaErrors( hipFree(energy_kev) );
	checkCudaErrors( hipFree(potential) );
	checkCudaErrors( hipFree(dw) );
	checkCudaErrors( hipFree(d_particles) );

	hipDeviceReset();

	return 0;
}
