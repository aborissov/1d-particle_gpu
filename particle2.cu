#define MAINFILE
#include <iostream>
#include <fstream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "hip/hip_runtime_api.h"

#include "constants.h"
#include "initial_conditions.h"
#include "evol.h"
#include "diagnostics.h"

using namespace std;

int main(int argc, char *argv[]){
	real *h_particles, *d_particles; // array of particles: 1d position, cosine of pitch angle, lorentz factor, exit time
	bool newflag = 1,newflag_trajectories = 1;
	float *dw;
	int threads_per_block = 32;
	int nblocks = nparticles/threads_per_block;  // make sure nparticles is a multiple of 32

	printf("nt %d dt %f Tfinal %f timeblocks %d\n ",nt,dt,Tfinal,timeblocks);
	printf("nblocks %d, threads_per_block %d threads %d, nparticles %d\n",nblocks,threads_per_block,threads_per_block*nblocks,nparticles);


	float milli;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	h_particles = (real *)malloc(nparticles*nfields*sizeof(real));
	
	checkCudaErrors( hipMalloc((void **)&d_energy_kev, nparticles*sizeof(real)) );
	checkCudaErrors( hipMalloc((void **)&d_potential, nparticles*sizeof(real)) );
	checkCudaErrors( hipMalloc((void **)&d_particles, nparticles*nfields*sizeof(real)) );
	checkCudaErrors( hipMalloc((void **)&dw, nparticles*nt*sizeof(float)) );
	initialise(h_particles);
	
	printf("size of: dw %d d_particles %d\n", nparticles*nt, nparticles*nfields);

	checkCudaErrors( hipMemcpy(d_particles,h_particles,nparticles*nfields*sizeof(real),hipMemcpyHostToDevice) );

	// random number generation
	
	hipEventRecord(start);
	
	hiprandGenerator_t gen;
	checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW) );
	checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
	checkCudaErrors( hiprandGenerateNormal(gen, dw, nparticles*nt, 0.0f, 1.0f) );
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);
	
	printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
	        milli, nparticles*nt/(0.001*milli));

	// run particles
	hipEventRecord(start);
	for (int j = 0; j < timeblocks; j++){
		checkCudaErrors( hiprandSetGeneratorOffset(gen, (unsigned long long) j*nparticles*nt-1) );
		checkCudaErrors( hiprandGenerateNormal(gen, dw, nparticles*nt, 0.0f, 1.0f) );
		move_particles<<<nblocks,threads_per_block>>>(d_particles,dw,d_energy_kev,d_potential,&j);
		getLastCudaError("move_particles execution failed\n");
		if (j % 100 == 0) printf("done timeblock %d\n",j);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);
	
	printf("kernel  execution time (ms): %f\n", milli);

	checkCudaErrors( hipMemcpy(h_particles,d_particles,nparticles*nfields*sizeof(real),hipMemcpyDeviceToHost) );
	write_particles(h_particles,newflag,argv[1]);
	cout << "size of particles array " << nparticles << endl;
	//for (int j = 0; j < nparticles; j++) printf("particle %d final energy %f position %f\n",j,(particles[nfields*j+2]-1)*511.0,particles[nfields*j]*Lscl);
	
	free(h_particles);
	checkCudaErrors( hipFree(d_energy_kev) );
	checkCudaErrors( hipFree(d_potential) );
	checkCudaErrors( hipFree(dw) );
	checkCudaErrors( hipFree(d_particles) );

	hipDeviceReset();

	return 0;
}
