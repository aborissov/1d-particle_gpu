#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>
#include "constants.h"

using namespace std;

__device__ real calc_vel(real gamma, real beta){
        real v = sqrt(c*c - c*c/(gamma*gamma));
        //cout << "gamma = " << gamma << endl;
        //cout << "beta = " << beta << endl;
        //cout << "v = " << v << endl;
        return v/Vscl;
}

__global__ void move_particles(real *particles, float *dw, real *energy_kev,real *potential){
        real E,J,eta,eta_spitzer,nu,kappa,lambda_ei = 2.0e8/Lscl,Epar_extent = 1.0e3;
		real beta,v,u,uperp,gamma,dudt,betadot,gammadot,dbeta,dgamma,position;
		int tid = threadIdx.x + blockIdx.x*blockDim.x;
		int random_index;
		random_index = tid;
		//random_index = timestep;

        for(int tstep = 0; tstep < nt; tstep++){
			position = particles[nfields*tid];
			beta = particles[nfields*tid+1];
			gamma = particles[nfields*tid+2];
        	v = calc_vel(gamma,beta);
        	u = v*gamma*beta;
        	uperp = v*gamma*sqrt(1.0-beta*beta);

			//printf("particle %d dw %f random_index %d\n",tid,dw[random_index],random_index);

			if (abs(position*Lscl) < Epar_extent){
				//kappa = eta_spitzer/eta;
				kappa = 1.0e-5;
			}
			else {
				J = 0;
				E = 0;
				if (particles[nfields*tid+3] == 0){
					particles[nfields*tid+3] = tstep*dt*Tscl;
					//cout << "particle " << tid << " " << timestep*dt*Tscl << endl;
				}
			}
			if (particles[nfields*tid+3] == 0){
				eta_spitzer = 2.4e3/(pow((double) Temp,1.5))/etascl;
				J = 1.0e4/Escl;		// NON-DIMENSIONAL!!! Note: ensures electric field of 10 V/m when eta = 10^-3 (non-dimensional)
				eta = 1.0e-3;		// NON-DIMENSIONAL!!!
    			E = eta*J;			// NON-DIMENSIONAL!!!

				if (abs(position*Lscl) < Epar_extent) nu = v/(lambda_ei*kappa);
				else nu = 0.0;
				//nu = 0.0;

        	        	dudt = q*E*Escl/m*Tscl/Vscl;
        	        	gammadot = u*Vscl/(c*c)*dudt*Vscl/(sqrt(1 + u*u*Vscl*Vscl/(c*c) + uperp*uperp*Vscl*Vscl/(c*c)));      // work in progress!!!
        	        	if (u == 0) betadot = 0;
        	        	else betadot = dudt/u*beta*(1.0-beta*beta);

        	        	dgamma = gammadot*dt;
        	        	dbeta = (betadot - beta*nu)*dt + sqrt((1.0 - beta*beta)*nu)*((real) dw[random_index]);
						random_index += blockDim.x;

        	        	beta += dbeta;
        	        	if (beta > 1.0){
							//cout << "beta = " << particles[nfields*tid+1] << endl;
							beta = -beta + floor(beta) + 1.0;
						}
        	        	else if (beta < -1.0){
							//cout << "beta = " << particles[nfields*tid+1] << endl;
							beta = -beta + ceil(beta) - 1.0;
						}
        	        	gamma += dgamma;
        	        	if (gamma < 1) gamma -= 2.0*dgamma;

        	        	v = calc_vel(gamma, beta);
        	        	position += beta*v*dt;
				
				energy_kev[tid] = (gamma-1.0)*511.0;
				potential[tid] = -eta*J*Escl*position*Lscl/1.0e3;
				

				particles[nfields*tid] = position;
				particles[nfields*tid+1] = beta;
				particles[nfields*tid+2] = gamma;
				
				//if (fabs(energy_kev[tid] - potential[tid] - energy_kev_0) < 1.0 && tid == 1){
				//	printf("particle %d deviated from energy conservation at time %f", tid, timestep*dt*Tscl);
				//	printf(" kinetic %f, potential %f, initial %f, difference %f\n", energy_kev[tid], potential[tid], energy_kev_0,energy_kev[tid] - potential[tid] - energy_kev_0);
				//	printf(" eta, J, Escl, position, epar_extent, %f %f %f %f %f\n", eta,J,Escl,position*Lscl,Epar_extent);
				//}
				if (fabs(energy_kev[tid] - potential[tid] - energy_kev_0) > 1.0){
					particles[nfields*tid] = Epar_extent/Lscl;
					printf("particle %d deviated from energy conservation at time %f", tid, tstep*dt*Tscl);
					printf(" kinetic %f, potential %f, initial %f, difference %f\n", energy_kev[tid], potential[tid], energy_kev_0,energy_kev[tid] - potential[tid] - energy_kev_0);
					printf(" eta, J, Escl, position, epar_extent, %f %f %f %f %f\n", eta,J,Escl,position*Lscl,Epar_extent);
				}
				//cout << "particle " << tid << " total energy " << energy_kev[tid] - potential[tid] - energy_kev_0  
				//	<< " kinetic, potential, initial " << energy_kev[tid] << " " << potential[tid] << " "  << energy_kev_0 << endl;
			}
        }
}

