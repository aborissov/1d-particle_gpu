#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>
#include "constants.h"

using namespace std;

__device__ real calc_vel(real gamma, real beta){
        real v = sqrt(c*c - c*c/(gamma*gamma));
        return v/Vscl;
}

__global__ void move_particles(real *particles, float *dw, real *energy_kev,real *potential, real *initial_energy_kev,int *timeblock){
        real E,J,eta,eta_spitzer,nu,kappa,lambda_ei = 2.0e8/Lscl,Epar_extent = 1.0e3, x_extent = 1.0e4;
		real beta,v,u,uperp,gamma,dudt,betadot,gammadot,dbeta,dgamma,position,t_final;
		int tid = threadIdx.x + blockIdx.x*blockDim.x;
		int random_index;
		random_index = threadIdx.x + nt*blockIdx.x*blockDim.x;

		position = particles[nfields*tid];
		beta = particles[nfields*tid+1];
		gamma = particles[nfields*tid+2];
		t_final = particles[nfields*tid+3];

        for(int tstep = 0; tstep < nt; tstep++){
        	v = calc_vel(gamma,beta);
        	u = v*gamma*beta;
        	uperp = v*gamma*sqrt(1.0-beta*beta);
				
			//eta_spitzer = 2.4e3/(pow((double) Temp,1.5))/etascl;
			eta_spitzer = 7.6e-8/etascl;	// corresponds to temperature 10^7 K
			J = 1.0e4/Escl;		// NON-DIMENSIONAL!!! Note: ensures electric field of 10 V/m when eta = 10^-3 (non-dimensional)
			eta = 1.0e-3;		// NON-DIMENSIONAL!!!
    		E = eta*J;			// NON-DIMENSIONAL!!!

			if (abs(position*Lscl) < Epar_extent) {
				kappa = 10000.0*eta_spitzer/eta;
				//kappa = 1.0e-8;

				//nu = v/(lambda_ei*kappa);
				//printf("Epar 1/nu, dt: %.12e, %.12e, eta_spitzer, eta, eta_spitzer/eta, %.12e,%.12e,%.12e\n",1.0/nu,dt,eta_spitzer,eta,eta_spitzer/eta);
				nu = 0.0;

        	    dudt = q*E*Escl/m*Tscl/Vscl;
        	    gammadot = u*Vscl/(c*c)*dudt*Vscl/(sqrt(1 + u*u*Vscl*Vscl/(c*c) + uperp*uperp*Vscl*Vscl/(c*c)));     
        	    if (u == 0) betadot = 0;
        	    else betadot = dudt/u*beta*(1.0-beta*beta);

        	    dgamma = gammadot*dt;
        	    dbeta = (betadot - beta*nu)*dt + sqrt((1.0 - beta*beta)*nu)*sqrt(dt)*((real) dw[random_index]);
			    random_index += blockDim.x;

        	    beta += dbeta;
        	    if (beta > 1.0){
					beta = -beta + floor(beta) + 1.0;
				}
        	    else if (beta < -1.0){
					beta = -beta + ceil(beta) - 1.0;
				}
        	    gamma += dgamma;
        	    if (gamma < 1) gamma -= 2.0*dgamma;

        	    v = calc_vel(gamma, beta);
        	    position += beta*v*dt;
				
				// checking for energy conservation
				energy_kev[tid] = (gamma-1.0)*m_keV;
				potential[tid] = -eta*J*Escl*position*Lscl/1.0e3;
				
				if (fabs(energy_kev[tid] - potential[tid] - initial_energy_kev[tid]) > 1.0){
					position = Epar_extent/Lscl;
					printf("particle %d deviated from energy conservation at time %f", tid, (t_final+dt)*Tscl);
					printf(" kinetic %f, potential %f, initial %f, difference %f\n", energy_kev[tid], potential[tid], initial_energy_kev[tid],energy_kev[tid] - potential[tid] - initial_energy_kev[tid]);
					printf(" eta, J, Escl, position, epar_extent, %f %f %f %f %f\n", eta,J,Escl,position*Lscl,Epar_extent);
				}
				t_final += dt;
			}
			else if (abs(position*Lscl) < x_extent) {
				//kappa = eta_spitzer/eta;
				kappa = 1.0e-6;

				nu = v/(lambda_ei*kappa);
				//printf("ne 1/nu, dt: %.12e, %.12e, kappa, %.12e\n",1.0/nu,dt_ne,kappa);
				//nu = 0.0;

        	    betadot = 0;

        	    dbeta = (betadot - beta*nu)*dt + sqrt((1.0 - beta*beta)*nu)*sqrt(dt)*((real) dw[random_index]);
			    random_index += blockDim.x;

        	    beta += dbeta;
        	    if (beta > 1.0){
					beta = -beta + floor(beta) + 1.0;
				}
        	    else if (beta < -1.0){
					beta = -beta + ceil(beta) - 1.0;
				}

        	    v = calc_vel(gamma, beta);
        	    position += beta*v*dt;
				if (abs(position*Lscl) < Epar_extent) {
					position -= beta*v*dt;
				}

				t_final += dt_ne;
				
			}
        }
		particles[nfields*tid] = position;
		particles[nfields*tid+1] = beta;
		particles[nfields*tid+2] = gamma;
		particles[nfields*tid+3] = t_final;
}

