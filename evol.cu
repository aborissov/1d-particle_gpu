#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>
#include "constants.h"

using namespace std;

__device__ real calc_vel(real gamma, real beta){
        real v = sqrt(c*c - c*c/(gamma*gamma));
        //cout << "gamma = " << gamma << endl;
        //cout << "beta = " << beta << endl;
        //cout << "v = " << v << endl;
        return v/Vscl;
}

__global__ void move_particles(real *particles, float *dw, real *energy_kev,real *potential){
        real E,J,eta,eta_spitzer,nu,kappa,lambda_ei = 2.0e8/Lscl,Epar_extent = 1.0e3;
		real beta,v,u,uperp,gamma,dudt,betadot,gammadot,dbeta,dgamma,position;
		int tid = threadIdx.x + blockIdx.x*blockDim.x;
		int random_index;
		random_index = threadIdx.x + nt*blockIdx.x*blockDim.x;
		//random_index = timestep;

        for(int tstep = 0; tstep < nt; tstep++){
			position = particles[nfields*tid];
			beta = particles[nfields*tid+1];
			gamma = particles[nfields*tid+2];
        	v = calc_vel(gamma,beta);
        	u = v*gamma*beta;
        	uperp = v*gamma*sqrt(1.0-beta*beta);

			//printf("particle %d dw %f random_index %d\n",tid,dw[random_index],random_index);

			if (abs(position*Lscl) > Epar_extent){
				J = 0;
				E = 0;
				if (particles[nfields*tid+3] == 0){
					particles[nfields*tid+3] = tstep*dt;
					//cout << "particle " << tid << " " << timestep*dt*Tscl << endl;
				}
			}
			else{
				//eta_spitzer = 2.4e3/(pow((double) Temp,1.5))/etascl;
				eta_spitzer = 7.6e-8/etascl;	// corresponds to temperature 10^7 K
				J = 1.0e4/Escl;		// NON-DIMENSIONAL!!! Note: ensures electric field of 10 V/m when eta = 10^-3 (non-dimensional)
				eta = 1.0e-3;		// NON-DIMENSIONAL!!!
    			E = eta*J;			// NON-DIMENSIONAL!!!
				kappa = eta_spitzer/eta;
				//kappa = 1.0e-5;

				nu = v/(lambda_ei*kappa);
				//nu = 0.0;

        	    dudt = q*E*Escl/m*Tscl/Vscl;
        	    gammadot = u*Vscl/(c*c)*dudt*Vscl/(sqrt(1 + u*u*Vscl*Vscl/(c*c) + uperp*uperp*Vscl*Vscl/(c*c)));     
        	    if (u == 0) betadot = 0;
        	    else betadot = dudt/u*beta*(1.0-beta*beta);

        	    dgamma = gammadot*dt;
        	    dbeta = (betadot - beta*nu)*dt + sqrt((1.0 - beta*beta)*nu)*sqrt(dt)*((real) dw[random_index]);
				random_index += blockDim.x;

        	    beta += dbeta;
        	    if (beta > 1.0){
					//cout << "beta = " << particles[nfields*tid+1] << endl;
					beta = -beta + floor(beta) + 1.0;
				}
        	    else if (beta < -1.0){
					//cout << "beta = " << particles[nfields*tid+1] << endl;
					beta = -beta + ceil(beta) - 1.0;
				}
        	    gamma += dgamma;
        	    if (gamma < 1) gamma -= 2.0*dgamma;

        	    v = calc_vel(gamma, beta);
        	    position += beta*v*dt;
				
				energy_kev[tid] = (gamma-1.0)*511.0;
				potential[tid] = -eta*J*Escl*position*Lscl/1.0e3;
				

				particles[nfields*tid] = position;
				particles[nfields*tid+1] = beta;
				particles[nfields*tid+2] = gamma;
				
				//if (fabs(energy_kev[tid] - potential[tid] - energy_kev_0) < 1.0 && tid == 1){
				//	printf("particle %d deviated from energy conservation at time %f", tid, timestep*dt*Tscl);
				//	printf(" kinetic %f, potential %f, initial %f, difference %f\n", energy_kev[tid], potential[tid], energy_kev_0,energy_kev[tid] - potential[tid] - energy_kev_0);
				//	printf(" eta, J, Escl, position, epar_extent, %f %f %f %f %f\n", eta,J,Escl,position*Lscl,Epar_extent);
				//}
				if (fabs(energy_kev[tid] - potential[tid] - energy_kev_0) > 1.0){
					particles[nfields*tid] = Epar_extent/Lscl;
					printf("particle %d deviated from energy conservation at time %f", tid, tstep*dt*Tscl);
					printf(" kinetic %f, potential %f, initial %f, difference %f\n", energy_kev[tid], potential[tid], energy_kev_0,energy_kev[tid] - potential[tid] - energy_kev_0);
					printf(" eta, J, Escl, position, epar_extent, %f %f %f %f %f\n", eta,J,Escl,position*Lscl,Epar_extent);
				}
				//printf("particle %d, total energy %f, of which kinetic %f, potential %f, initial %f\n", tid,  energy_kev[tid] - potential[tid] - energy_kev_0, energy_kev[tid],  potential[tid], energy_kev_0);
			}
        }
}

